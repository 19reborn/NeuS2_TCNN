#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are permitted
 * provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright notice, this list of
 *       conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright notice, this list of
 *       conditions and the following disclaimer in the documentation and/or other materials
 *       provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
 *       to endorse or promote products derived from this software without specific prior written
 *       permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
 * STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *//*
 */

/** @file   cutlass_mlp.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  CUTLASS implementation of an optimized multi-layer perceptron. Supports online training
 *          and simultaneous inference.
 */

#include <tiny-cuda-nn/networks/cutlass_mlp.h>

#include <tiny-cuda-nn/cutlass_matmul.h>

TCNN_NAMESPACE_BEGIN

template <typename T>
__global__ void set_constant_value_view_vector(
	const uint32_t n_elements,
	const uint32_t n_pos_dim,
	const T value,
	tcnn::MatrixView<T> output
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t elem_idx = i / n_pos_dim;
	const uint32_t dim_idx = i - elem_idx * n_pos_dim;

	output(dim_idx, elem_idx) = value;
}

template <typename T>
__global__ void matrix_multiple(
	const uint32_t n_elements,
	const uint32_t row,
	const uint32_t col,
	const uint32_t batch,
	tcnn::MatrixView<T> back,
	tcnn::MatrixView<T> front,
	tcnn::MatrixView<T> output
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t row_id = i / col;
	const uint32_t col_id = i - row_id * col;
	for (uint32_t j = 0; j < batch; j++){
		output(row_id, col_id) += back(row_id,j) * front(col_id, j);
		printf("row:%d,col:%d,output,back,front:%f,%f,%f\n",row_id,col_id,(float)output(row_id, col_id),(float)back(row_id,j),(float)front(col_id, j));
	}

}

template <typename T>
__global__ void debug_log(
	const uint32_t  n_elements,
	tcnn::MatrixView<T> output
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	printf("%d: %.10f\n",i, (float)output(i,0));
}

template <typename T>
CutlassMLP<T>::CutlassMLP(
	uint32_t input_width,
	uint32_t network_width,
	uint32_t output_width,
	uint32_t n_hidden_layers,
	Activation activation,
	Activation output_activation
) :
m_input_width{input_width},
m_network_width{network_width},
m_output_width{output_width},
m_n_hidden_layers{n_hidden_layers},
m_activation{activation},
m_output_activation{output_activation},
m_can_fuse_activation{activation != Activation::Sine}
{
	m_padded_output_width = next_multiple(m_output_width, REQUIRED_ALIGNMENT());

	if (n_hidden_layers > 0) {
		m_n_hidden_matmuls = n_hidden_layers-1;
	} else {
		m_n_hidden_matmuls = 0;
	}

	// Create matrices related to weights
	if (n_hidden_layers == 0) {
		m_weight_matrices.emplace_back(nullptr, m_padded_output_width, m_input_width);
		m_weight_matrices_inference.emplace_back(nullptr, m_padded_output_width, m_input_width);
		m_weight_matrices_full_precision.emplace_back(nullptr, m_padded_output_width, m_input_width);
		m_gradient_matrices.emplace_back(nullptr, m_padded_output_width, m_input_width);

		m_weight_matrices_stored.emplace_back(nullptr, m_padded_output_width, m_input_width);
		m_weight_matrices_inference_stored.emplace_back(nullptr, m_padded_output_width, m_input_width);
		m_gradient_matrices_stored.emplace_back(nullptr, m_padded_output_width, m_input_width);
	} else {
		m_weight_matrices.emplace_back(nullptr, m_network_width, m_input_width);
		m_weight_matrices_inference.emplace_back(nullptr, m_network_width, m_input_width);
		m_weight_matrices_full_precision.emplace_back(nullptr, m_network_width, m_input_width);
		m_gradient_matrices.emplace_back(nullptr, m_network_width, m_input_width);

		m_weight_matrices_stored.emplace_back(nullptr, m_network_width, m_input_width);
		m_weight_matrices_inference_stored.emplace_back(nullptr, m_network_width, m_input_width);
		m_gradient_matrices_stored.emplace_back(nullptr, m_network_width, m_input_width);

		for (uint32_t i = 0; i < m_n_hidden_matmuls; ++i) {
			m_weight_matrices.emplace_back(nullptr, m_network_width, m_network_width);
			m_weight_matrices_inference.emplace_back(nullptr, m_network_width, m_network_width);
			m_weight_matrices_full_precision.emplace_back(nullptr, m_network_width, m_network_width);
			m_gradient_matrices.emplace_back(nullptr, m_network_width, m_network_width);

			m_weight_matrices_stored.emplace_back(nullptr, m_network_width, m_network_width);
			m_weight_matrices_inference_stored.emplace_back(nullptr, m_network_width, m_network_width);
			m_gradient_matrices_stored.emplace_back(nullptr, m_network_width, m_network_width);
		}

		m_weight_matrices.emplace_back(nullptr, m_padded_output_width, m_network_width);
		m_weight_matrices_inference.emplace_back(nullptr, m_padded_output_width, m_network_width);
		m_weight_matrices_full_precision.emplace_back(nullptr, m_padded_output_width, m_network_width);
		m_gradient_matrices.emplace_back(nullptr, m_padded_output_width, m_network_width);

		m_weight_matrices_stored.emplace_back(nullptr, m_padded_output_width, m_network_width);
		m_weight_matrices_inference_stored.emplace_back(nullptr, m_padded_output_width, m_network_width);
		m_gradient_matrices_stored.emplace_back(nullptr, m_padded_output_width, m_network_width);
	}

	// Determine total number of memory entries and set it
	m_total_n_params = 0;
	for (const auto& m : m_weight_matrices) {
		m_total_n_params += m.n_elements();
	}

	// 1 stream per matrix.
	m_training_splitk_streams.resize(m_n_hidden_layers + 1);
	m_training_splitk_events.resize(m_n_hidden_layers + 1);

	for (size_t i = 0; i < m_training_splitk_streams.size(); ++i) {
		CUDA_CHECK_THROW(hipStreamCreate(&m_training_splitk_streams[i]));
		CUDA_CHECK_THROW(hipEventCreate(&m_training_splitk_events[i]));
	}
}

template <typename T>
CutlassMLP<T>::~CutlassMLP() {
	for (size_t i = 0; i < m_training_splitk_streams.size(); ++i) {
		free_gpu_memory_arena(m_training_splitk_streams[i]);

		CUDA_CHECK_PRINT(hipEventDestroy(m_training_splitk_events[i]));
		CUDA_CHECK_PRINT(hipStreamDestroy(m_training_splitk_streams[i]));
	}
}

template <typename CutlassLayer, typename T>
bool compute_layer(
	hipStream_t stream,
	bool is_inference,
	Activation activation,
	const GPUMatrix<T, RM>& weights,
	const GPUMatrixDynamic<T>& input,
	GPUMatrixDynamic<T>& output,
	GPUMatrixDynamic<T>& activation_output
) {
	bool can_fuse_activation = true;
	if (!is_inference) {
		// Never disallow fusing if the caller passes the same output and activation_output buffers... in that case,
		// invertibility of the activation function may be ignored.
		can_fuse_activation &= activation != Activation::Sine || &output == &activation_output;
		// can_fuse_activation = false;
	}

	if (can_fuse_activation) {
		fc_multiply<CutlassLayer>(stream, weights, input, output, activation);
	} else {
		fc_multiply<CutlassLayer>(stream, weights, input, output);
		activation_gpu(stream, activation, output, activation_output);
	}

	return can_fuse_activation;
	// return true;
}

template <typename CutlassLayer, typename T>
bool compute_inference_layer(
	hipStream_t stream,
	Activation activation,
	const GPUMatrix<T, RM>& weights,
	const GPUMatrixDynamic<T>& input,
	GPUMatrixDynamic<T>& output
) {
	return compute_layer<CutlassLayer>(stream, true, activation, weights, input, output, output);
}

template <typename T>
void CutlassMLP<T>::inference_mixed_precision_impl(hipStream_t stream, const GPUMatrixDynamic<T>& input, GPUMatrixDynamic<T>& output, bool use_inference_params) {
	// If there are no hidden layers, the network is just a simple matmul.
	if (m_n_hidden_layers == 0) {
		compute_inference_layer<LastLayer>(stream, m_output_activation, input_weight_matrix(use_inference_params), input, output);
		return;
	}

	uint32_t batch_size = input.n();
	GPUMatrix<T> inference_tmp[2] = {
		GPUMatrix<T>{m_network_width, batch_size, stream},
		GPUMatrix<T>{m_network_width, batch_size, stream},
	};

	m_inference_graph.capture_and_execute(stream, false, [&]() {
		// Run the actual network
		{
			uint32_t tmp_idx = 0;

			// Input layer
			compute_inference_layer<FullLayer>(stream, m_activation, input_weight_matrix(use_inference_params), input, inference_tmp[tmp_idx++ % 2]);

			// Hidden layers
			for (uint32_t i = 0; i < m_n_hidden_matmuls; ++i) {
				compute_inference_layer<FullLayer>(stream, m_activation, weight_matrix_at(use_inference_params, i), inference_tmp[(tmp_idx + 1) % 2], inference_tmp[tmp_idx % 2]);
				++tmp_idx;
			}

			// Output
			compute_inference_layer<LastLayer>(stream, m_output_activation, output_weight_matrix(use_inference_params), inference_tmp[(tmp_idx + 1) % 2], output);
		}
	});
}

template <typename T>
std::unique_ptr<Context> CutlassMLP<T>::forward_impl(hipStream_t stream, const GPUMatrixDynamic<T>& input, GPUMatrixDynamic<T>* output, bool use_inference_params, bool prepare_input_gradients) {
	// If there are no hidden layers, the network is just a simple matmul. No tmp buffers required
	if (m_n_hidden_layers == 0) {
		if (output) {
			compute_layer<LastLayer>(stream, false, m_output_activation, input_weight_matrix(use_inference_params), input, *output, *output);
			// compute_layer<LastLayer>(stream, true, m_output_activation, input_weight_matrix(use_inference_params), input, *output, *output);
		}
		return std::make_unique<ForwardContext>(); // Nothing to save -- empty context
	}

	// Make sure our temporary buffers have the correct size for the given batch size
	uint32_t batch_size = input.n();
	auto forward = allocate_forward_buffers(stream, batch_size);

	// Run the actual network
	uint32_t tmp_idx = 0;

	bool fused = compute_layer<FullLayer>(
		stream,
		false,
		m_activation,
		input_weight_matrix(use_inference_params),
		input,
		// forward->hidden_input.at(tmp_idx),
		forward->hidden.at(tmp_idx),
		m_can_fuse_activation ? forward->hidden.at(tmp_idx) : forward->hidden.at(tmp_idx+1)
	);
	tmp_idx += fused ? 1 : 2;

	// layers
	for (uint32_t i = 0; i < m_n_hidden_matmuls; ++i) {
		fused = compute_layer<FullLayer>(
			stream,
			false,
			m_activation,
			weight_matrix_at(use_inference_params, i),
			forward->hidden.at(tmp_idx-1),
			// forward->hidden_input.at(tmp_idx),
			// forward->hidden.at(tmp_idx)
			forward->hidden.at(tmp_idx),
			m_can_fuse_activation ? forward->hidden.at(tmp_idx) : forward->hidden.at(tmp_idx+1)
		);
		tmp_idx += fused ? 1 : 2;
	}

	if (output) {
		compute_layer<LastLayer>(stream, false, m_output_activation, output_weight_matrix(use_inference_params), forward->hidden.at(tmp_idx-1), *output, *output);
		// compute_layer<LastLayer>(stream, true, m_output_activation, output_weight_matrix(use_inference_params), forward->hidden.at(tmp_idx-1), *output, *output);
	}

	return forward;
}

template <typename T>
void CutlassMLP<T>::backward_impl(
	hipStream_t stream,
	const Context& ctx,
	const GPUMatrixDynamic<T>& input,
	const GPUMatrixDynamic<T>& output,
	const GPUMatrixDynamic<T>& dL_doutput,
	GPUMatrixDynamic<T>* dL_dinput,
	bool use_inference_params,
	EGradientMode param_gradients_mode
) {
	// Make sure our temporary buffers have the correct size for the given batch size
	uint32_t batch_size = dL_doutput.n();

	std::vector<GPUMatrix<T>> backward_tmp(num_forward_activations());
	for (uint32_t i = 0; i < num_forward_activations(); ++i) {
		backward_tmp[i] = GPUMatrix<T>{m_network_width, batch_size, stream};
	}

	// Compute transfer of output activation in-place... it's treated specially for performance reasons
	GPUMatrixDynamic<T> backward_output_tmp;
	if (m_output_activation != Activation::None) {
		backward_output_tmp = {m_padded_output_width, batch_size, stream, dL_doutput.layout()};
		activation_backward_output_gpu(stream, dL_doutput.n_elements(), m_output_activation, output.data(), dL_doutput.data(), backward_output_tmp.data());
		// activation_backward_output_gpu: stream, input_elements, activation, activation_value, input_value, output_value)
	}

	// Backprop
	// - weight_gradient.T = activation * output_gradient.T
	// - input_gradient = weights.T * output_gradient
	// - RELU: pre_activation_gradinet = post_activation_gradient if val > 0 else 0

	const float param_gradient_beta = param_gradients_mode == EGradientMode::Accumulate ? 1.0f : 0.0f;

	{
		const auto& forward = dynamic_cast<const ForwardContext&>(ctx);

		int split_k_factor = batch_size / std::min((uint32_t)(1 << 12), batch_size);

		const GPUMatrixDynamic<T>& tmp_dL_doutput = m_output_activation == Activation::None ? dL_doutput : backward_output_tmp;

		// If there are no hidden layers, the network is just a simple matmul
		if (m_n_hidden_layers == 0) {
			if (param_gradients_mode != EGradientMode::Ignore) {
				hipEventRecord(m_training_splitk_events.at(0), stream);
				hipStreamWaitEvent(m_training_splitk_streams.at(0), m_training_splitk_events.at(0), 0);

				// Compute weight gradients
				fc_multiply_split_k<LastLayerK>(m_training_splitk_streams.at(0), tmp_dL_doutput, input.transposed(), input_gradient_matrix(), split_k_factor, param_gradient_beta);

				hipEventRecord(m_training_splitk_events.at(0), m_training_splitk_streams.at(0));
			}

			if (dL_dinput) {
				fc_multiply<FullLayer>(stream, input_weight_matrix(use_inference_params).transposed(), tmp_dL_doutput, *dL_dinput);
			}

			if (param_gradients_mode != EGradientMode::Ignore) {
				hipStreamWaitEvent(stream, m_training_splitk_events.at(0), 0);
			}
			return;
		}

		uint32_t tmp_idx = (m_can_fuse_activation ? (m_n_hidden_matmuls+1) : ((m_n_hidden_matmuls+1) * 2)) - 1;
		uint32_t backward_tmp_idx = 0;

		if (param_gradients_mode != EGradientMode::Ignore) {
			// Output layer
			hipEventRecord(m_training_splitk_events.at(backward_tmp_idx), stream);
			hipStreamWaitEvent(m_training_splitk_streams.at(backward_tmp_idx), m_training_splitk_events.at(backward_tmp_idx), 0);

			// Compute weight gradients
			fc_multiply_split_k<LastLayerK>(m_training_splitk_streams.at(backward_tmp_idx), tmp_dL_doutput, forward.hidden.at(tmp_idx).transposed(), output_gradient_matrix(), split_k_factor, param_gradient_beta);

			hipEventRecord(m_training_splitk_events.at(backward_tmp_idx), m_training_splitk_streams.at(backward_tmp_idx));
		}

		if (!m_can_fuse_activation) {
			fc_multiply<FullLayer>(stream, output_weight_matrix(use_inference_params).transposed(), tmp_dL_doutput, backward_tmp.at(backward_tmp_idx));
			activation_backward_gpu(stream, m_activation, forward.hidden.at(tmp_idx-1), backward_tmp.at(backward_tmp_idx));
		} else {
			fc_multiply<FullLayer>(stream, output_weight_matrix(use_inference_params).transposed(), tmp_dL_doutput, forward.hidden.at(tmp_idx), backward_tmp.at(backward_tmp_idx), m_activation, true);
		}

		tmp_idx -= m_can_fuse_activation ? 1 : 2;
		++backward_tmp_idx;

		// layers
		for (uint32_t i = 0; i < m_n_hidden_matmuls; ++i) {
			uint32_t matrix_idx = m_n_hidden_matmuls - i - 1;

			if (param_gradients_mode != EGradientMode::Ignore) {
				hipEventRecord(m_training_splitk_events.at(backward_tmp_idx), stream);
				hipStreamWaitEvent(m_training_splitk_streams.at(backward_tmp_idx), m_training_splitk_events.at(backward_tmp_idx), 0);
				fc_multiply_split_k<FullLayerK>(m_training_splitk_streams.at(backward_tmp_idx), backward_tmp.at(backward_tmp_idx-1), forward.hidden.at(tmp_idx).transposed(), gradient_matrix_at(matrix_idx), split_k_factor, param_gradient_beta);
				hipEventRecord(m_training_splitk_events.at(backward_tmp_idx), m_training_splitk_streams.at(backward_tmp_idx));
			}

			if (!m_can_fuse_activation) {
				fc_multiply<FullLayer>(stream, weight_matrix_at(use_inference_params, matrix_idx).transposed(), backward_tmp.at(backward_tmp_idx-1), backward_tmp.at(backward_tmp_idx));
				activation_backward_gpu(stream, m_activation, forward.hidden.at(tmp_idx-1), backward_tmp.at(backward_tmp_idx));
			} else {
				fc_multiply<FullLayer>(stream, weight_matrix_at(use_inference_params, matrix_idx).transposed(), backward_tmp.at(backward_tmp_idx-1), forward.hidden.at(tmp_idx), backward_tmp.at(backward_tmp_idx), m_activation, true);
			}

			tmp_idx -= m_can_fuse_activation ? 1 : 2;
			++backward_tmp_idx;
		}

		if (param_gradients_mode != EGradientMode::Ignore) {
			hipEventRecord(m_training_splitk_events.at(backward_tmp_idx), stream);
			hipStreamWaitEvent(m_training_splitk_streams.at(backward_tmp_idx), m_training_splitk_events.at(backward_tmp_idx), 0);
			fc_multiply_split_k<FullLayerK>(m_training_splitk_streams.at(backward_tmp_idx), backward_tmp.at(backward_tmp_idx-1), input.transposed(), input_gradient_matrix(), split_k_factor, param_gradient_beta);
			hipEventRecord(m_training_splitk_events.at(backward_tmp_idx), m_training_splitk_streams.at(backward_tmp_idx));
		}

		// If requested, compute sensitivity of loss w.r.t. inputs
		if (dL_dinput) {
			// optimization opportunity to only compute sensitivity w.r.t selected SUBSET of inputs. Useful for NFs, where conditional dims stay the same.
			fc_multiply<FullLayer>(stream, input_weight_matrix(use_inference_params).transposed(), backward_tmp.at(backward_tmp_idx-1), *dL_dinput);
		}
	}

	if (param_gradients_mode != EGradientMode::Ignore) {
		// All the per-layer split-k matrix multiplications summing over
		// the batch are computed in parallel streams to the actual
		// backpropagation. Here, we need to wait for all of these to complete.
		for (auto& event : m_training_splitk_events) {
			hipStreamWaitEvent(stream, event, 0);
		}
	}
}

// Assume that derivative is a square matrix. n_row == n_col.
template <typename T, bool SET_ROW = true>
__global__ void apply_relu_to_derivative_by_forward(
	const uint32_t n_elements,
	const uint32_t n_row,
	const uint32_t n_col,
	const tcnn::MatrixView<T> forward,
	tcnn::MatrixView<T> derivative
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t i_row = i / n_row;
	if (forward(i_row, 0) < T(0)) {
		const uint32_t i_col = i - n_row * i_row;
		if (SET_ROW) {
			derivative(i_row, i_col) = T(0);
		}
		else {
			derivative(i_col, i_row) = T(0);
		}
	}
}

template <typename T, bool SET_ROW = true>
__global__ void apply_relu_to_derivative_by_forward_batch(
	const uint32_t n_elements,
	const uint32_t n_pos_dim,
	const tcnn::MatrixView<T> forward,
	tcnn::MatrixView<T> derivative
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t elem_idx = i / n_pos_dim;
	const uint32_t dim_idx = i - elem_idx * n_pos_dim;
	// printf("apply relu:%f!\n",(float)forward(dim_idx, elem_idx));
	// printf("elem_idx:%d,dim_idx:%d,value: %f!\n",elem_idx,dim_idx,(float)forward(dim_idx, elem_idx));
	if (forward(dim_idx, elem_idx) < T(0)) {
	// if (forward(dim_idx, elem_idx) <= T(0)) {
		// printf("elem_idx:%d,dim_idx:%d\n",elem_idx,dim_idx);
		derivative(dim_idx, elem_idx) = T(0);
	}
}

template <typename T>
__global__ void kernel_compute_update_weight(
	const uint32_t n_elements,
	const uint32_t n_row,
	const uint32_t n_col,
	const tcnn::MatrixView<T> tmp_matrix_front,
	const tcnn::MatrixView<T> tmp_matrix_back,
	tcnn::MatrixView<T> update_weight,
	EGradientMode param_gradients_mode
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t i_row = i / n_row;
	const uint32_t i_col = i - n_row * i_row;

	T tmp_matrix_front_sum = 0.0f;
	T tmp_matrix_back_sum = 0.0f;
	for (uint32_t k = 0; k < n_row; ++k) {
		tmp_matrix_front_sum += tmp_matrix_front(k, i_col);
	}
	for (uint32_t l = 0; l < n_col; ++l) {
		tmp_matrix_back_sum += tmp_matrix_back(i_row, l);
	}
	// printf("tmp_matrix_back_sum %f\n",tmp_matrix_back_sum);
	// printf("tmp_matrix_front_sum %f\n",tmp_matrix_front_sum);
	if (param_gradients_mode == EGradientMode::Accumulate){
		update_weight(i_row, i_col) += tmp_matrix_front_sum * tmp_matrix_back_sum;
	}
	else if (param_gradients_mode == EGradientMode::Overwrite){
		update_weight(i_row, i_col) = tmp_matrix_front_sum * tmp_matrix_back_sum;
	}
	// printf("update_weight: %f", update_weight(i_row, i_col));
}

// Implemented by Yiming Wang <w752531540@gmail.com>
// According to Equation 8,9 in NeuS2 <https://arxiv.org/abs/2212.05231>
template <typename T>
void CutlassMLP<T>::backward_backward_input_impl(
	hipStream_t stream,
	const Context& ctx,
	const GPUMatrixDynamic<T>& input,
	const GPUMatrixDynamic<T>& dL_ddLdinput,
	const GPUMatrixDynamic<T>& dL_doutput,
	GPUMatrixDynamic<T>* dL_ddLdoutput,
	GPUMatrixDynamic<T>* dL_dinput,
	bool use_inference_params,
	EGradientMode param_gradients_mode
) {

	// there exists m_hidden_layers + 1 layers in total.
	// Make sure our temporary buffers have the correct size for the given batch size
	uint32_t batch_size = dL_doutput.n();

	uint32_t num_tmp_matrix = m_n_hidden_layers + 3; // since that there exist one more default output layer
	std::vector<GPUMatrix<T>> tmp_front_multiply(num_tmp_matrix);

	for (uint32_t i = 1; i < num_tmp_matrix - 2; ++i) {
		uint32_t matrix_size = full_weight_matrix_at(use_inference_params, i - 1).m();
		tmp_front_multiply[i].set_size_unsafe(matrix_size, batch_size);
	}
	auto tmp_front_multiply_alloc = GPUMatrixBase::allocate_shared_memory(stream, tmp_front_multiply);
	tmp_front_multiply[0] = GPUMatrix<T>{dL_ddLdinput.data(),input_weight_matrix(use_inference_params).n(),batch_size};
	
	std::vector<GPUMatrix<T>> tmp_back_multiply(num_tmp_matrix);
	
	for (uint32_t i = num_tmp_matrix - 2; i > 1; --i) {
		uint32_t matrix_size = full_weight_matrix_at(use_inference_params, i-1).n();
		tmp_back_multiply[i].set_size_unsafe(matrix_size, batch_size);
	}
	auto tmp_back_multiply_alloc = GPUMatrixBase::allocate_shared_memory(stream, tmp_back_multiply);
	tmp_back_multiply[num_tmp_matrix-1] = GPUMatrix<T>{dL_doutput.data(),output_weight_matrix(use_inference_params).m(),batch_size};
	
	const float param_gradient_beta = param_gradients_mode == EGradientMode::Accumulate ? 1.0f : 0.0f;

	{
		const auto& forward = dynamic_cast<const ForwardContext&>(ctx);

		int split_k_factor = batch_size / std::min((uint32_t)(1 << 12), batch_size);
		
		for (uint32_t i = 1; i < num_tmp_matrix - 2; ++i) {
			auto& cur_tmp_front_multiply = tmp_front_multiply.at(i);
			fc_multiply<FullLayer>(stream, 
				full_weight_matrix_at(use_inference_params, i-1),
				tmp_front_multiply.at(i-1), 
				forward.hidden.at(i-1), 
				cur_tmp_front_multiply, 
				m_activation, true);

		}

		for (uint32_t i = num_tmp_matrix - 2; i > 1; --i) {
			auto& cur_tmp_back_multiply = tmp_back_multiply.at(i);
			fc_multiply<FullLayer>(stream, 
				full_weight_matrix_at(use_inference_params, i-1).transposed(),
				tmp_back_multiply.at(i+1),
				forward.hidden.at(i-2),  
				cur_tmp_back_multiply, 
				m_activation, true);

		}
	
		for (uint32_t i = 1; i < num_tmp_matrix - 1; ++i) {
			auto& gradient_matrix = m_gradient_matrices.at(i-1);
			if (param_gradients_mode != EGradientMode::Ignore) {
				hipEventRecord(m_training_splitk_events.at(i-1), stream);
				hipStreamWaitEvent(m_training_splitk_streams.at(i-1), m_training_splitk_events.at(i-1), 0);
				fc_multiply_split_k<FullLayerK>(m_training_splitk_streams.at(i-1), tmp_back_multiply.at(i+1), tmp_front_multiply.at(i-1).transposed(), gradient_matrix, split_k_factor, param_gradient_beta);
				hipEventRecord(m_training_splitk_events.at(i-1), m_training_splitk_streams.at(i-1));
			}
		}

	}

	if (param_gradients_mode != EGradientMode::Ignore) {

		for (auto& event : m_training_splitk_events) {
			hipStreamWaitEvent(stream, event, 0);
		}
	}

}

template <typename T>
std::unique_ptr<typename CutlassMLP<T>::ForwardContext> CutlassMLP<T>::allocate_forward_buffers(hipStream_t stream, uint32_t batch_size) {
	auto forward = std::make_unique<ForwardContext>();

	forward->hidden.resize(num_forward_activations());
	for (uint32_t i = 0; i < num_forward_activations(); ++i) {
		forward->hidden[i] = GPUMatrix<T>{m_network_width, batch_size, stream};
	}

	return forward;
}

template <typename T>
void CutlassMLP<T>::set_params(T* params, T* inference_params, T* backward_params, T* gradients) {
	size_t current_pos = 0;
	for (size_t i = 0; i < m_weight_matrices.size(); ++i) {
		m_weight_matrices[i].set_data_unsafe(params + current_pos);
		m_weight_matrices_inference[i].set_data_unsafe(inference_params + current_pos);
		m_gradient_matrices[i].set_data_unsafe(gradients + current_pos);
		current_pos += m_weight_matrices[i].n_elements();
	}
}

template <typename T>
void CutlassMLP<T>::initialize_params(pcg32& rnd, float* params_full_precision, T* params, T* inference_params, T* backward_params, T* gradients, float scale) {
	set_params(params, inference_params, backward_params, gradients);

	size_t current_pos = 0;
	for (size_t i = 0; i < m_weight_matrices_full_precision.size(); ++i) {
		m_weight_matrices_full_precision[i].set_data_unsafe(params_full_precision + current_pos);
		current_pos += m_weight_matrices_full_precision[i].n_elements();

		if (m_activation == Activation::Sine) {
			if (i == 0) {
				m_weight_matrices_full_precision[i].initialize_siren_uniform_first(rnd, scale);
			} else {
				m_weight_matrices_full_precision[i].initialize_siren_uniform(rnd, scale);
			}
		} else {
			m_weight_matrices_full_precision[i].initialize_xavier_uniform(rnd, scale);
		}
	}
}

template <typename T>
void CutlassMLP<T>::read_params(std::vector<T*> params, std::vector<T*> inference_params, std::vector<T*> backward_params, std::vector<T*> gradients){

	for (size_t i = 0; i < m_weight_matrices.size(); ++i) {
		
		m_weight_matrices_stored[i].set_data_unsafe(m_weight_matrices[i].data());
		m_weight_matrices_inference_stored[i].set_data_unsafe(m_weight_matrices_inference[i].data());
		m_gradient_matrices_stored[i].set_data_unsafe(m_gradient_matrices[i].data());
	}

	return;
}


template <typename T>
void CutlassMLP<T>::set_params_from_matrix(std::vector<T*> params, std::vector<T*> inference_params, std::vector<T*> backward_params, std::vector<T*> gradients){

	for (size_t i = 0; i < m_weight_matrices.size(); ++i) {

		m_weight_matrices[i].set_data_unsafe(m_weight_matrices_stored[i].data());
		m_weight_matrices_inference[i].set_data_unsafe(m_weight_matrices_inference_stored[i].data());
		m_gradient_matrices[i].set_data_unsafe(m_gradient_matrices_stored[i].data());
	}

	return;
}

// Explicitly instantiate CutlassMLP classes.
template class CutlassMLP<network_precision_t>;

TCNN_NAMESPACE_END
